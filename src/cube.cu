#include "hip/hip_runtime.h"

#include <cmath>
#include <fstream>
#include <iostream>
#include <vector>

#include "dir-enum.hh"
#include "geometry3.hh"
#include "plane.hh"
#include "trop-enum.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


void allocate() {
  std::cout << __PRETTY_FUNCTION__ << std::endl;
  // hipMalloc((void**)&n_x_d, sizeof(int));
  // hipMalloc((void**)&n_y_d, sizeof(int));
  // hipMalloc((void**)&n_z_d, sizeof(int));
  // hipMalloc((void**)&field_d, size() * sizeof(coord3d));
  // hipMalloc((void**)&origin_d, sizeof(coord3d));
  // hipMalloc((void**)&spacing_d, sizeof(coord3d));
}


void upload() {
  std::cout << __PRETTY_FUNCTION__ << std::endl;
  // hipMemcpy(n_x_d, &n_x, sizeof(int), hipMemcpyHostToDevice);
  // hipMemcpy(n_y_d, &n_y, sizeof(int), hipMemcpyHostToDevice);
  // hipMemcpy(n_z_d, &n_z, sizeof(int), hipMemcpyHostToDevice);
  // hipMemcpy(field_d, field.data(), size() * sizeof(coord3d), hipMemcpyHostToDevice);
  // hipMemcpy(origin_d, &origin, sizeof(coord3d), hipMemcpyHostToDevice);
  // hipMemcpy(spacing_d, &spacing, sizeof(coord3d), hipMemcpyHostToDevice);
}


void download() {
  std::cout << __PRETTY_FUNCTION__ << std::endl;
  // CudaMemcpy( n_x_d, &n_x, sizeof(int) , hipMemcpyDeviceToHost);
  // CudaMemcpy( n_y_d, &n_y, sizeof(int) , hipMemcpyDeviceToHost);
  // CudaMemcpy( n_z_d, &n_z, sizeof(int) , hipMemcpyDeviceToHost);
  // CudaMemcpy( field_d, field.data(), size() * sizeof(coord3d) , hipMemcpyDeviceToHost);
  // CudaMemcpy( origin_d, &origin,  sizeof(coord3d) , hipMemcpyDeviceToHost);
  // CudaMemcpy( spacing_d, &spacing,  sizeof(coord3d) , hipMemcpyDeviceToHost);
}

void deallocate() {
  std::cout << __PRETTY_FUNCTION__ << std::endl;
  //hipFree(n_x_d);
  //hipFree(n_y_d);
  //hipFree(n_z_d);
  // hipFree(field_d);
  //hipFree(origin_d);
  //hipFree(spacing_d);
}


__global__ void classify_points_kernel(Tropicity* res_d, coord3d* field_d, int n_x, int n_y, int n_z) {
  // std::cout << __PRETTY_FUNCTION__ << std::endl;
 printf("hello from the gpu\n");
}


std::vector<Tropicity> classify_points_cudax(const std::vector<coord3d>& field, int nx, int ny, int nz, coord3d origin, coord3d spacing,
                                             const std::vector<coord3d>& coords, Direction bfielddir) {
  int64_t n = coords.size();
  std::vector<Tropicity> res(n);
  coord3d* field_d;
  Tropicity* res_d;

  hipMalloc((void**)&field_d, n * sizeof(coord3d));
  hipMalloc((void**)&res_d, n * sizeof(Tropicity));

  hipMemcpy(field_d, field.data(), n * sizeof(coord3d), hipMemcpyHostToDevice);

  classify_points_kernel<<<3, 3>>>(res_d, field_d, nx, ny, nz);

  hipMemcpy(res.data(), res_d, n * sizeof(Tropicity), hipMemcpyDeviceToHost);

  hipFree(field_d);
  hipFree(res_d);
  return res;
}


