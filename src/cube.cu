#include "hip/hip_runtime.h"
#include <cmath>
#include <fstream>
#include <iostream>
#include <vector>

#include "dir-enum.hh"
#include "geometry3_d.hh"
#include "plane.hh"
#include "trop-enum.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
// #include <vector_functions.h>


__device__ inline double3 operator*(const double d, const double3 d3) {
  const double xval = d * d3.x;
  const double yval = d * d3.y;
  const double zval = d * d3.z;
  return make_double3(xval, yval, zval);
}

__device__ inline double3 operator+(const double3 d3a, const double3 d3b) {
  const double xval = d3a.x * d3b.x;
  const double yval = d3a.y * d3b.y;
  const double zval = d3a.z * d3b.z;
  return make_double3(xval, yval, zval);
}


__device__ bool outofbounds(const coord3d_d& pos, int64_t nx, int64_t ny, int64_t nz) {
  if (pos.x[0] >= nx - 1 || pos.x[1] >= ny - 1 || pos.x[2] >= nz - 1 ||
      pos.x[0] < 0 || pos.x[1] < 0 || pos.x[2] < 0) {
    return true;
  }
  return false;
}


// trilinear interpolation
__device__ bool getvector_v1(const coord3d_d& pos, const hipPitchedPtr field_d, const int64_t nx, const int64_t ny, const int64_t nz, coord3d_d& res_vec) {
  // int indx = threadIdx.x + blockIdx.x * blockDim.x;
  // if (indx < 2) {
  //  printf("x %f, y %f, z %f \n", pos[0], pos[1], pos[2]);
  // }

  char* ptr = (char*)field_d.ptr;
  // pitch being xdim * sizeof(thing) rounded up to a multiple of 32
  size_t pitch = field_d.pitch;

  if (outofbounds(pos, nx, ny, nz))
    return false;

  double x = pos[0];
  double y = pos[1];
  double z = pos[2];
  int x0 = int(floor(pos[0]));
  int x1 = x0 + 1;
  int y0 = int(floor(pos[1]));
  int y1 = y0 + 1;
  int z0 = int(floor(pos[2]));
  int z1 = z0 + 1;
  coord3d_d v000 = *(coord3d_d*)(ptr + (pitch * ny * z0 + pitch * y0 + x0 * sizeof(coord3d_d)));
  coord3d_d v001 = *(coord3d_d*)(ptr + (pitch * ny * z1 + pitch * y0 + x0 * sizeof(coord3d_d)));
  coord3d_d v010 = *(coord3d_d*)(ptr + (pitch * ny * z0 + pitch * y1 + x0 * sizeof(coord3d_d)));
  coord3d_d v011 = *(coord3d_d*)(ptr + (pitch * ny * z1 + pitch * y1 + x0 * sizeof(coord3d_d)));
  coord3d_d v100 = *(coord3d_d*)(ptr + (pitch * ny * z0 + pitch * y0 + x1 * sizeof(coord3d_d)));
  coord3d_d v101 = *(coord3d_d*)(ptr + (pitch * ny * z1 + pitch * y0 + x1 * sizeof(coord3d_d)));
  coord3d_d v110 = *(coord3d_d*)(ptr + (pitch * ny * z0 + pitch * y1 + x1 * sizeof(coord3d_d)));
  coord3d_d v111 = *(coord3d_d*)(ptr + (pitch * ny * z1 + pitch * y1 + x1 * sizeof(coord3d_d)));
  coord3d_d aux0 = (x1 - x) * v000 + (x - x0) * v100;
  coord3d_d aux1 = (x1 - x) * v010 + (x - x0) * v110;
  coord3d_d aux2 = (x1 - x) * v001 + (x - x0) * v101;
  coord3d_d aux3 = (x1 - x) * v011 + (x - x0) * v111;
  coord3d_d aux4 = (y1 - y) * aux0 + (y - y0) * aux1;
  coord3d_d aux5 = (y1 - y) * aux2 + (y - y0) * aux3;
  res_vec = (z1 - z) * aux4 + (z - z0) * aux5;
  return true;
}


__device__ inline coord3d_d get_val_v2(const int x, const int y, const int z,
                                       hipTextureObject_t field_x, hipTextureObject_t field_y, hipTextureObject_t field_z) {
  int2 xvalint2 = tex3D<int2>(field_x, x, y, z);
  double xval = __hiloint2double(xvalint2.y, xvalint2.x);
  int2 yvalint2 = tex3D<int2>(field_y, x, y, z);
  double yval = __hiloint2double(yvalint2.y, yvalint2.x);
  int2 zvalint2 = tex3D<int2>(field_z, x, y, z);
  double zval = __hiloint2double(zvalint2.y, zvalint2.x);
  return coord3d_d(xval, yval, zval);
}


// trilinear interpolation
__device__ bool getvector_v2(const coord3d_d& pos,
                             hipTextureObject_t field_x, hipTextureObject_t field_y, hipTextureObject_t field_z,
                             const int64_t nx, const int64_t ny, const int64_t nz, coord3d_d& res_vec) {
  // int indx = threadIdx.x + blockIdx.x * blockDim.x;
  // if (indx < 2) {
  //  printf("x %f, y %f, z %f \n", pos[0], pos[1], pos[2]);
  // }

  if (outofbounds(pos, nx, ny, nz))
    return false;

  double x = pos[0];
  double y = pos[1];
  double z = pos[2];
  int x0 = int(floor(pos[0]));
  int x1 = x0 + 1;
  int y0 = int(floor(pos[1]));
  int y1 = y0 + 1;
  int z0 = int(floor(pos[2]));
  int z1 = z0 + 1;
  coord3d_d v000 = get_val_v2(x0, y0, z0, field_x, field_y, field_z);
  coord3d_d v001 = get_val_v2(x0, y0, z1, field_x, field_y, field_z);
  coord3d_d v010 = get_val_v2(x0, y1, z0, field_x, field_y, field_z);
  coord3d_d v011 = get_val_v2(x0, y1, z1, field_x, field_y, field_z);
  coord3d_d v100 = get_val_v2(x1, y0, z0, field_x, field_y, field_z);
  coord3d_d v101 = get_val_v2(x1, y0, z1, field_x, field_y, field_z);
  coord3d_d v110 = get_val_v2(x1, y1, z0, field_x, field_y, field_z);
  coord3d_d v111 = get_val_v2(x1, y1, z1, field_x, field_y, field_z);
  coord3d_d aux0 = (x1 - x) * v000 + (x - x0) * v100;
  coord3d_d aux1 = (x1 - x) * v010 + (x - x0) * v110;
  coord3d_d aux2 = (x1 - x) * v001 + (x - x0) * v101;
  coord3d_d aux3 = (x1 - x) * v011 + (x - x0) * v111;
  coord3d_d aux4 = (y1 - y) * aux0 + (y - y0) * aux1;
  coord3d_d aux5 = (y1 - y) * aux2 + (y - y0) * aux3;
  res_vec = (z1 - z) * aux4 + (z - z0) * aux5;
  return true;
}


// trilinear interpolation
__device__ inline bool getvector_v3(const coord3d_d& pos, hipTextureObject_t field_d,
                                    const int64_t nx, const int64_t ny, const int64_t nz, coord3d_d& res_vec) {
  // int indx = threadIdx.x + blockIdx.x * blockDim.x;
  // if (indx < 2) {
  //  printf("x %f, y %f, z %f \n", pos[0], pos[1], pos[2]);
  // }

  if (outofbounds(pos, nx, ny, nz))
    return false;

  double x = pos[0];
  double y = pos[1];
  double z = pos[2];

  float4 val = tex3D<float4>(field_d, x, y, z);
  res_vec = coord3d_d(val.x, val.y, val.z);
  return true;
}


// Runge-Kutta method, 4th order
// c --> positions, k --> vectors at c
__device__ bool extend_rungekutta_v1(const hipPitchedPtr field_d, const int64_t nx, const int64_t ny, const int64_t nz,
                                     const coord3d_d& prevpos, float step_length, coord3d_d& newpos) {
  // int indx = threadIdx.x + blockIdx.x * blockDim.x;
  coord3d_d c0 = prevpos;
  coord3d_d k0;
  bool good = getvector_v1(c0, field_d, nx, ny, nz, k0);
  k0 = k0.normalised() * step_length;

  const coord3d_d c1 = c0 + k0 * 0.5;
  coord3d_d k1;
  good = getvector_v1(c1, field_d, nx, ny, nz, k1);
  if (!good)
    return false;
  k1 = k1.normalised() * step_length;

  const coord3d_d c2 = c0 + k1 * 0.5;
  coord3d_d k2;
  good = getvector_v1(c2, field_d, nx, ny, nz, k2);
  if (!good)
    return false;
  k2 = k2.normalised() * step_length;

  const coord3d_d c3 = c0 + k2;
  coord3d_d k3;
  good = getvector_v1(c3, field_d, nx, ny, nz, k3);
  if (!good)
    return false;
  k3 = k3.normalised() * step_length;

  const coord3d_d c4 = c0 + (k0 + k1 * 2.0 + k2 * 2.0 + k3) / 6.0;
  coord3d_d k4;
  good = getvector_v1(c4, field_d, nx, ny, nz, k4);
  if (!good)
    return false;
  newpos = c4;
  return true;
}


// Runge-Kutta method, 4th order
// c --> positions, k --> vectors at c
__device__ bool extend_rungekutta_v2(const hipTextureObject_t field_x, const hipTextureObject_t field_y, const hipTextureObject_t field_z,
                                     const int64_t nx, const int64_t ny, const int64_t nz,
                                     const coord3d_d& prevpos, float step_length, coord3d_d& newpos) {
  // int indx = threadIdx.x + blockIdx.x * blockDim.x;
  coord3d_d c0 = prevpos;
  coord3d_d k0;
  bool good = getvector_v2(c0, field_x, field_y, field_z, nx, ny, nz, k0);
  k0 = k0.normalised() * step_length;

  const coord3d_d c1 = c0 + k0 * 0.5;
  coord3d_d k1;
  good = getvector_v2(c1, field_x, field_y, field_z, nx, ny, nz, k1);
  if (!good)
    return false;
  k1 = k1.normalised() * step_length;

  const coord3d_d c2 = c0 + k1 * 0.5;
  coord3d_d k2;
  good = getvector_v2(c2, field_x, field_y, field_z, nx, ny, nz, k2);
  if (!good)
    return false;
  k2 = k2.normalised() * step_length;

  const coord3d_d c3 = c0 + k2;
  coord3d_d k3;
  good = getvector_v2(c3, field_x, field_y, field_z, nx, ny, nz, k3);
  if (!good)
    return false;
  k3 = k3.normalised() * step_length;

  const coord3d_d c4 = c0 + (k0 + k1 * 2.0 + k2 * 2.0 + k3) / 6.0;
  coord3d_d k4;
  good = getvector_v2(c4, field_x, field_y, field_z, nx, ny, nz, k4);
  if (!good)
    return false;
  newpos = c4;
  return true;
}


// Runge-Kutta method, 4th order
// c --> positions, k --> vectors at c
__device__ bool extend_rungekutta_v3(const hipTextureObject_t field_d,
                                     const int64_t nx, const int64_t ny, const int64_t nz,
                                     const coord3d_d& prevpos, float step_length, coord3d_d& newpos) {
  // int indx = threadIdx.x + blockIdx.x * blockDim.x;
  coord3d_d c0 = prevpos;
  coord3d_d k0;
  bool good = getvector_v3(c0, field_d, nx, ny, nz, k0);
  k0 = k0.normalised() * step_length;

  const coord3d_d c1 = c0 + k0 * 0.5;
  coord3d_d k1;
  good = getvector_v3(c1, field_d, nx, ny, nz, k1);
  if (!good)
    return false;
  k1 = k1.normalised() * step_length;

  const coord3d_d c2 = c0 + k1 * 0.5;
  coord3d_d k2;
  good = getvector_v3(c2, field_d, nx, ny, nz, k2);
  if (!good)
    return false;
  k2 = k2.normalised() * step_length;

  const coord3d_d c3 = c0 + k2;
  coord3d_d k3;
  good = getvector_v3(c3, field_d, nx, ny, nz, k3);
  if (!good)
    return false;
  k3 = k3.normalised() * step_length;

  const coord3d_d c4 = c0 + (k0 + k1 * 2.0 + k2 * 2.0 + k3) / 6.0;
  coord3d_d k4;
  good = getvector_v3(c4, field_d, nx, ny, nz, k4);
  if (!good)
    return false;
  newpos = c4;
  return true;
}


__device__ void complete_trajectory_v1(const hipPitchedPtr field_d, const int nx, const int ny, const int nz,
                                       coord3d_d* __restrict__ positions, int& index, int max_points_traj,
                                       float return_ratio, float step_length, bool& out_of_bounds) {
  // const int indx = threadIdx.x + blockIdx.x * blockDim.x;
  out_of_bounds = false;
  double dist2farthest = -1; // if this is set at 0 at declaration, the following while loop will never run
  if (index > 0) {
    for (int i = 0; i <= index; i++)
      dist2farthest = std::max(dist2farthest, (positions[i] - positions[0]).norm());
  }

  // if we get to a point that is less than return_ratio of the longest distance in the trajectory
  while ((positions[index] - positions[0]).norm() > return_ratio * dist2farthest) {
    if (!extend_rungekutta_v1(field_d, nx, ny, nz,
                              positions[index], step_length, positions[index + 1])) {
      out_of_bounds = true;
      // printf("%d: %d oob\n", indx, index);
      return;
    }
    index++;

    dist2farthest = std::max(dist2farthest, (positions[index] - positions[0]).norm());

    if (index == max_points_traj - 2) {
      step_length *= 1.5;
      index = 0;
      dist2farthest = -1;
    }
  }
  // printf("%d: %d\n", indx, index);
}


__device__ void complete_trajectory_v2(const hipTextureObject_t field_x, const hipTextureObject_t field_y, const hipTextureObject_t field_z,
                                       const int nx, const int ny, const int nz,
                                       coord3d_d* __restrict__ positions, int& index, int max_points_traj,
                                       float return_ratio, float step_length, bool& out_of_bounds) {
  // const int indx = threadIdx.x + blockIdx.x * blockDim.x;
  out_of_bounds = false;
  double dist2farthest = -1; // if this is set at 0 at declaration, the following while loop will never run
  if (index > 0) {
    for (int i = 0; i <= index; i++)
      dist2farthest = std::max(dist2farthest, (positions[i] - positions[0]).norm());
  }

  // if we get to a point that is less than return_ratio of the longest distance in the trajectory
  while ((positions[index] - positions[0]).norm() > return_ratio * dist2farthest) {
    if (!extend_rungekutta_v2(field_x, field_y, field_z, nx, ny, nz,
                              positions[index], step_length, positions[index + 1])) {
      out_of_bounds = true;
      // printf("%d: %d oob\n", indx, index);
      return;
    }
    index++;

    dist2farthest = std::max(dist2farthest, (positions[index] - positions[0]).norm());

    if (index == max_points_traj - 2) {
      step_length *= 1.5;
      index = 0;
      dist2farthest = -1;
    }
  }
  // printf("%d: %d\n", indx, index);
}


__device__ void complete_trajectory_v3(const hipTextureObject_t field_d,
                                       const int nx, const int ny, const int nz,
                                       coord3d_d* __restrict__ positions, int& index, int max_points_traj,
                                       float return_ratio, float step_length, bool& out_of_bounds) {
  // const int indx = threadIdx.x + blockIdx.x * blockDim.x;
  out_of_bounds = false;
  double dist2farthest = -1; // if this is set at 0 at declaration, the following while loop will never run
  if (index > 0) {
    for (int i = 0; i <= index; i++)
      dist2farthest = std::max(dist2farthest, (positions[i] - positions[0]).norm());
  }

  // if we get to a point that is less than return_ratio of the longest distance in the trajectory
  while ((positions[index] - positions[0]).norm() > return_ratio * dist2farthest) {
    if (!extend_rungekutta_v3(field_d, nx, ny, nz,
                              positions[index], step_length, positions[index + 1])) {
      out_of_bounds = true;
      // printf("%d: %d oob\n", indx, index);
      return;
    }
    index++;

    dist2farthest = std::max(dist2farthest, (positions[index] - positions[0]).norm());

    if (index == max_points_traj - 2) {
      step_length *= 1.5;
      index = 0;
      dist2farthest = -1;
    }
  }
  // printf("%d: %d\n", indx, index);
}


__device__ Tropicity classify_trajectory(const coord3d_d* __restrict__ positions, int n_points_in_traj, Direction bfielddir, bool out_of_bounds) {
  // int indx = threadIdx.x + blockIdx.x * blockDim.x;
  // printf("p in traj %d: %d\n", indx, n_points_in_traj);
  coord3d_d bfield;
  switch (bfielddir) {
    case Direction::pos_x: {
      bfield = coord3d_d(1, 0, 0);
      break;
    }
    case Direction::neg_x: {
      bfield = coord3d_d(-1, 0, 0);
      break;
    }
    case Direction::pos_y: {
      bfield = coord3d_d(0, 1, 0);
      break;
    }
    case Direction::neg_y: {
      bfield = coord3d_d(0, -1, 0);
      break;
    }
    case Direction::pos_z: {
      bfield = coord3d_d(0, 0, 1);
      break;
    }
    case Direction::neg_z: {
      bfield = coord3d_d(0, 0, -1);
      break;
    }
    default: {
      return Tropicity::input_error;
    }
  }

  if (out_of_bounds)
    return Tropicity::outofbounds;

  coord3d_d crosssum(0, 0, 0);
  for (size_t i = 0; i < n_points_in_traj; i++) {
    crosssum += positions[(i - 1 + n_points_in_traj) % n_points_in_traj].cross(positions[i]);
  }
  // crossum += positions[positions.size()-1].cross(positions[0]);
  // if (indx < 40) printf("cross: %f/%f/%f\n", crosssum[0], crosssum[1], crosssum[2]);

  double dot_product = bfield.dot(crosssum);
  if (dot_product > 0)
    return Tropicity::paratropic;
  else if (dot_product < 0)
    return Tropicity::diatropic;
  else
    return Tropicity::unclassifyable;
}


__global__ void classify_points_kernel_v1(coord3d_d* __restrict__ points, int64_t n_points,
                                          const hipPitchedPtr field_d, const int64_t nx, const int64_t ny, const int64_t nz,
                                          coord3d_d* __restrict__ trajectories_d, float step_length, int64_t max_points_traj,
                                          Direction bfielddir, Tropicity* __restrict__ tropicities_d) {
  const int32_t indx = threadIdx.x + blockIdx.x * blockDim.x;
  // if (indx < 2) printf("hello from the gpu: %d\n", indx);

  if (indx > n_points - 1)
    return;

  coord3d_d vec(0, 0, 0);
  // if (indx < 2) printf("pos %d %f/%f/%f\n", indx, points[indx][0], points[indx][1], points[indx][2]);
  bool good = getvector_v1(points[indx], field_d, nx, ny, nz, vec);
  // if (indx < 2) printf("found vec %d %d: %f/%f/%f\n", indx, good, vec[0], vec[1], vec[2]);
  if (!good) {
    tropicities_d[indx] = Tropicity::outofbounds;
    return;
  }

  bool out_of_bounds;
  int current_index_in_traj = 0;
  float return_ratio = 0.2;
  trajectories_d[max_points_traj * indx] = points[indx];
  complete_trajectory_v1(field_d, nx, ny, nz,
                         trajectories_d + max_points_traj * indx, current_index_in_traj, max_points_traj,
                         return_ratio, step_length, out_of_bounds);
  tropicities_d[indx] = classify_trajectory(trajectories_d + max_points_traj * indx, current_index_in_traj + 1, bfielddir, out_of_bounds);
}


__global__ void classify_points_kernel_v2(coord3d_d* __restrict__ points, int64_t n_points,
                                          hipTextureObject_t field_x, hipTextureObject_t field_y, hipTextureObject_t field_z,
                                          const int64_t nx, const int64_t ny, const int64_t nz,
                                          coord3d_d* __restrict__ trajectories_d, float step_length, int64_t max_points_traj,
                                          Direction bfielddir, Tropicity* __restrict__ tropicities_d) {

  const int32_t indx = threadIdx.x + blockIdx.x * blockDim.x;
  // if (indx < 2) printf("hello from the gpu: %d\n", indx);

  if (indx > n_points - 1)
    return;

  coord3d_d vec(0, 0, 0);
  // if (indx < 2) printf("pos %d %f/%f/%f\n", indx, points[indx][0], points[indx][1], points[indx][2]);
  bool good = getvector_v2(points[indx], field_x, field_y, field_z, nx, ny, nz, vec);
  // if (indx < 2) printf("found vec %d %d: %f/%f/%f\n", indx, good, vec[0], vec[1], vec[2]);
  if (!good) {
    tropicities_d[indx] = Tropicity::outofbounds;
    return;
  }

  bool out_of_bounds;
  int current_index_in_traj = 0;
  float return_ratio = 0.2;
  trajectories_d[max_points_traj * indx] = points[indx];
  complete_trajectory_v2(field_x, field_y, field_z, nx, ny, nz,
                         trajectories_d + max_points_traj * indx, current_index_in_traj, max_points_traj,
                         return_ratio, step_length, out_of_bounds);
  tropicities_d[indx] = classify_trajectory(trajectories_d + max_points_traj * indx, current_index_in_traj + 1, bfielddir, out_of_bounds);
}


__global__ void classify_points_kernel_v3(coord3d_d* __restrict__ points, int64_t n_points,
                                          hipTextureObject_t field_d,
                                          const int64_t nx, const int64_t ny, const int64_t nz,
                                          coord3d_d* __restrict__ trajectories_d, float step_length, int64_t max_points_traj,
                                          Direction bfielddir, Tropicity* __restrict__ tropicities_d) {
  const int32_t indx = threadIdx.x + blockIdx.x * blockDim.x;
  // if (indx < 2) printf("hello from the gpu: %d\n", indx);

  if (indx > n_points - 1) {
    return;
  }

  coord3d_d vec(0, 0, 0);
  // if (indx < 2) printf("pos %d %f/%f/%f\n", indx, points[indx][0], points[indx][1], points[indx][2]);
  bool good = getvector_v3(points[indx], field_d, nx, ny, nz, vec);
  // if (indx < 2) printf("found vec %d %d: %f/%f/%f\n", indx, good, vec[0], vec[1], vec[2]);
  if (!good) {
    tropicities_d[indx] = Tropicity::outofbounds;
    return;
  }

  bool out_of_bounds;
  int current_index_in_traj = 0;
  float return_ratio = 0.2;
  trajectories_d[max_points_traj * indx] = points[indx];
  complete_trajectory_v3(field_d, nx, ny, nz,
                         trajectories_d + max_points_traj * indx, current_index_in_traj, max_points_traj,
                         return_ratio, step_length, out_of_bounds);
  tropicities_d[indx] = classify_trajectory(trajectories_d + max_points_traj * indx, current_index_in_traj + 1, bfielddir, out_of_bounds);
}


std::vector<Tropicity> classify_points_cudax_v1(const double* field_a, const int64_t nx, const int64_t ny, const int64_t nz, double* origin_a, double* spacing_a,
                                                const double* start_points_a, int64_t n_points, Direction bfielddir) {
  // std::cout << __PRETTY_FUNCTION__ << std::endl;
#if 0
  float steplength = 0.01;
#else
  float step_length_ratio = 0.05;
  float step_length = step_length_ratio * spacing_a[0];
#endif
  int64_t max_points_traj = 10000;

  std::vector<Tropicity> res(n_points);
  coord3d_d* field = new coord3d_d[nx * ny * nz];
  coord3d_d* start_points = new coord3d_d[n_points];

  coord3d_d* start_points_d;
  Tropicity* res_d;
  coord3d_d* trajectories_d;

  for (int64_t i = 0; i < nx * ny * nz; i++)
    for (int64_t j = 0; j < 3; j++)
      field[i][j] = field_a[3 * i + j];
  for (int64_t i = 0; i < n_points; i++)
    for (int64_t j = 0; j < 3; j++)
      start_points[i][j] = start_points_a[3 * i + j];

  hipPitchedPtr field_d;
  hipExtent field_extent = make_hipExtent(nx * sizeof(coord3d_d), ny, nz);
  hipMalloc3D(&field_d, field_extent);

  hipMemcpy3DParms memCopyParameters = {0};
  memCopyParameters.srcPtr = make_hipPitchedPtr(field, nx * sizeof(coord3d_d), ny, nz);
  memCopyParameters.dstPtr = field_d;
  memCopyParameters.extent = field_extent;
  memCopyParameters.kind = hipMemcpyHostToDevice;

  hipMemcpy3DAsync(&memCopyParameters, 0);
  // printf("nx, ny, nz %lu, %lu, %lu\n", field_d.pitch, field_d.xsize, field_d.ysize);

  // alloc
  hipMalloc((void**)&start_points_d, n_points * sizeof(coord3d_d));
  hipMalloc((void**)&trajectories_d, n_points * max_points_traj * sizeof(coord3d_d));
  hipMalloc((void**)&res_d, n_points * sizeof(Tropicity));

  // copy to device
  hipMemcpy(start_points_d, start_points, n_points * sizeof(coord3d_d), hipMemcpyHostToDevice);
  // cout << "e " << hipGetLastError() << endl;

  int block_size = 256;
  int grid_size = n_points / block_size + (n_points % block_size != 0);
  // std::cout << "points / gridsize / blocksize: " << n_points << ", " << grid_size << ", " << block_size << std::endl;
  classify_points_kernel_v1<<<grid_size, block_size>>>(start_points_d, n_points,
                                                       field_d, nx, ny, nz,
                                                       trajectories_d, step_length, max_points_traj,
                                                       bfielddir, res_d);
  // cout << "e " << hipGetLastError() << endl;

  // copy from device
  hipMemcpy(res.data(), res_d, n_points * sizeof(Tropicity), hipMemcpyDeviceToHost);

  // dealloc
  hipFree(field_d.ptr);
  hipFree(start_points_d);
  hipFree(trajectories_d);
  hipFree(res_d);
  delete[] field;
  delete[] start_points;
  return res;
}


std::vector<Tropicity> classify_points_cudax_v2(double* field_x_a, double* field_y_a, double* field_z_a,
                                                const int64_t nx, const int64_t ny, const int64_t nz, double* origin_a, double* spacing_a,
                                                const double* start_points_a, int64_t n_points, Direction bfielddir) {
  // std::cout << __PRETTY_FUNCTION__ << std::endl;
#if 0
  float steplength = 0.01;
#else
  float step_length_ratio = 0.05;
  float step_length = step_length_ratio * spacing_a[0];
#endif
  int64_t max_points_traj = 10000;

  std::vector<Tropicity> res(n_points);
  coord3d_d* start_points = new coord3d_d[n_points];

  coord3d_d* start_points_d;
  Tropicity* res_d;
  coord3d_d* trajectories_d;

  for (int64_t i = 0; i < n_points; i++)
    for (int64_t j = 0; j < 3; j++)
      start_points[i][j] = start_points_a[3 * i + j];

  hipArray_t field_x_d, field_y_d, field_z_d;
  // hipChannelFormatDesc desc = hipCreateChannelDesc<double>();
  hipChannelFormatDesc desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindSigned); // we pretend to store int2 instead of double
  hipExtent field_extent = make_hipExtent(nx, ny, nz);
  hipMalloc3DArray(&field_x_d, &desc, field_extent);
  hipMalloc3DArray(&field_y_d, &desc, field_extent);
  hipMalloc3DArray(&field_z_d, &desc, field_extent);

  hipMemcpy3DParms memCopyParametersX = {0};
  memCopyParametersX.srcPtr = make_hipPitchedPtr(field_x_a, nx * sizeof(double), nx, ny);
  memCopyParametersX.dstArray = field_x_d;
  memCopyParametersX.extent = field_extent;
  memCopyParametersX.kind = hipMemcpyHostToDevice;
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  hipMemcpy3DParms memCopyParametersY = {0};
  memCopyParametersY.srcPtr = make_hipPitchedPtr(field_y_a, nx * sizeof(double), nx, ny);
  memCopyParametersY.dstArray = field_y_d;
  memCopyParametersY.extent = field_extent;
  memCopyParametersY.kind = hipMemcpyHostToDevice;
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  hipMemcpy3DParms memCopyParametersZ = {0};
  memCopyParametersZ.srcPtr = make_hipPitchedPtr(field_z_a, nx * sizeof(double), nx, ny);
  memCopyParametersZ.dstArray = field_z_d;
  memCopyParametersZ.extent = field_extent;
  memCopyParametersZ.kind = hipMemcpyHostToDevice;
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  hipMemcpy3DAsync(&memCopyParametersX, 0);
  hipMemcpy3DAsync(&memCopyParametersY, 0);
  hipMemcpy3DAsync(&memCopyParametersZ, 0);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;
  // printf("nx, ny, nz %lu, %lu, %lu\n", field_d.pitch, field_d.xsize, field_d.ysize);

  // prepare textures
  struct hipResourceDesc fieldXResDesc;
  memset(&fieldXResDesc, 0, sizeof(fieldXResDesc));
  fieldXResDesc.resType = hipResourceTypeArray;
  fieldXResDesc.res.array.array = field_x_d;
  struct hipResourceDesc fieldYResDesc;
  memset(&fieldYResDesc, 0, sizeof(fieldYResDesc));
  fieldYResDesc.resType = hipResourceTypeArray;
  fieldYResDesc.res.array.array = field_y_d;
  struct hipResourceDesc fieldZResDesc;
  memset(&fieldZResDesc, 0, sizeof(fieldZResDesc));
  fieldZResDesc.resType = hipResourceTypeArray;
  fieldZResDesc.res.array.array = field_z_d;

  struct hipTextureDesc fieldTexDesc;
  memset(&fieldTexDesc, 0, sizeof(fieldTexDesc));
  fieldTexDesc.addressMode[0] = hipAddressModeBorder; // alternatively: wrap, clamp, mirror
  fieldTexDesc.addressMode[1] = hipAddressModeBorder; // alternatively: wrap, clamp, mirror
  fieldTexDesc.addressMode[2] = hipAddressModeBorder; // alternatively: wrap, clamp, mirror
  fieldTexDesc.filterMode = hipFilterModePoint;       // ie, interpolate linearly
  fieldTexDesc.readMode = hipReadModeElementType;
  fieldTexDesc.normalizedCoords = 0;

  hipTextureObject_t fieldXTexture = 0, fieldYTexture = 0, fieldZTexture = 0;
  hipCreateTextureObject(&fieldXTexture, &fieldXResDesc, &fieldTexDesc, nullptr);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;
  hipCreateTextureObject(&fieldYTexture, &fieldYResDesc, &fieldTexDesc, nullptr);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;
  hipCreateTextureObject(&fieldZTexture, &fieldZResDesc, &fieldTexDesc, nullptr);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;


  // alloc
  hipMalloc((void**)&start_points_d, n_points * sizeof(coord3d_d));
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;
  hipMalloc((void**)&trajectories_d, n_points * max_points_traj * sizeof(coord3d_d));
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;
  hipMalloc((void**)&res_d, n_points * sizeof(Tropicity));
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  // copy to device
  hipMemcpy(start_points_d, start_points, n_points * sizeof(coord3d_d), hipMemcpyHostToDevice);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  int block_size = 256;
  int grid_size = n_points / block_size + (n_points % block_size != 0);
  // std::cout << "points / gridsize / blocksize: " << n_points << ", " << grid_size << ", " << block_size << std::endl;
  classify_points_kernel_v2<<<grid_size, block_size>>>(start_points_d, n_points,
                                                       fieldXTexture, fieldYTexture, fieldZTexture, nx, ny, nz,
                                                       trajectories_d, step_length, max_points_traj,
                                                       bfielddir, res_d);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  // copy from device
  hipMemcpy(res.data(), res_d, n_points * sizeof(Tropicity), hipMemcpyDeviceToHost);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  hipDestroyTextureObject(fieldXTexture);
  hipDestroyTextureObject(fieldYTexture);
  hipDestroyTextureObject(fieldZTexture);

  // dealloc
  hipFree(start_points_d);
  hipFree(trajectories_d);
  hipFree(res_d);
  delete[] start_points;
  return res;
}


std::vector<Tropicity> classify_points_cudax_v3(float* field_x_a, float* field_y_a, float* field_z_a,
                                                const int64_t nx, const int64_t ny, const int64_t nz, double* origin_a, double* spacing_a,
                                                const double* start_points_a, int64_t n_points, Direction bfielddir) {
  // std::cout << __PRETTY_FUNCTION__ << std::endl;
#if 0
  float steplength = 0.01;
#else
  float step_length_ratio = 0.05;
  float step_length = step_length_ratio * spacing_a[0];
#endif
  int64_t max_points_traj = 10000;

  std::vector<Tropicity> res(n_points);
  coord3d_d* start_points = new coord3d_d[n_points];

  coord3d_d* start_points_d;
  Tropicity* res_d;
  coord3d_d* trajectories_d;

  for (int64_t i = 0; i < n_points; i++)
    for (int64_t j = 0; j < 3; j++)
      start_points[i][j] = start_points_a[3 * i + j];

  float4* field_float4 = new float4[nx * ny * nz];
  for (int64_t i = 0; i < nx * ny * nz; i++) {
    field_float4[i].x = field_x_a[i];
    field_float4[i].y = field_y_a[i];
    field_float4[i].z = field_z_a[i];
  }

  hipArray_t field_d;
  hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
  // hipChannelFormatDesc desc = hipCreateChannelDesc(32, 32, 32, 0, hipChannelFormatKindFloat);
  hipExtent field_extent = make_hipExtent(nx, ny, nz);
  hipMalloc3DArray(&field_d, &desc, field_extent);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  hipMemcpy3DParms memCopyParameters = {0};
  memCopyParameters.srcPtr = make_hipPitchedPtr(field_float4, nx * sizeof(float4), nx, ny);
  memCopyParameters.dstArray = field_d;
  memCopyParameters.extent = field_extent;
  memCopyParameters.kind = hipMemcpyHostToDevice;
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  hipMemcpy3DAsync(&memCopyParameters, 0);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;
  // printf("nx, ny, nz %lu, %lu, %lu\n", field_d.pitch, field_d.xsize, field_d.ysize);

  // prepare textures
  struct hipResourceDesc fieldResDesc;
  memset(&fieldResDesc, 0, sizeof(fieldResDesc));
  fieldResDesc.resType = hipResourceTypeArray;
  fieldResDesc.res.array.array = field_d;

  struct hipTextureDesc fieldTexDesc;
  memset(&fieldTexDesc, 0, sizeof(fieldTexDesc));
  fieldTexDesc.addressMode[0] = hipAddressModeBorder; // alternatively: wrap, clamp, mirror
  fieldTexDesc.addressMode[1] = hipAddressModeBorder; // alternatively: wrap, clamp, mirror
  fieldTexDesc.addressMode[2] = hipAddressModeBorder; // alternatively: wrap, clamp, mirror
  fieldTexDesc.filterMode = hipFilterModeLinear;      // ie, interpolate linearly
  fieldTexDesc.readMode = hipReadModeElementType;
  fieldTexDesc.normalizedCoords = 0;

  hipTextureObject_t fieldTexture = 0;
  hipCreateTextureObject(&fieldTexture, &fieldResDesc, &fieldTexDesc, nullptr);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;


  // alloc
  hipMalloc((void**)&start_points_d, n_points * sizeof(coord3d_d));
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;
  hipMalloc((void**)&trajectories_d, n_points * max_points_traj * sizeof(coord3d_d));
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;
  hipMalloc((void**)&res_d, n_points * sizeof(Tropicity));
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  // copy to device
  hipMemcpy(start_points_d, start_points, n_points * sizeof(coord3d_d), hipMemcpyHostToDevice);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  int block_size = 256;
  int grid_size = n_points / block_size + (n_points % block_size != 0);
  // std::cout << "points / gridsize / blocksize: " << n_points << ", " << grid_size << ", " << block_size << std::endl;
  classify_points_kernel_v3<<<grid_size, block_size>>>(start_points_d, n_points,
                                                       fieldTexture, nx, ny, nz,
                                                       trajectories_d, step_length, max_points_traj,
                                                       bfielddir, res_d);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  // copy from device
  hipMemcpy(res.data(), res_d, n_points * sizeof(Tropicity), hipMemcpyDeviceToHost);
  // cout << "e " << hipGetErrorName(hipGetLastError()) << endl;

  hipDestroyTextureObject(fieldTexture);

  // dealloc
  hipFree(start_points_d);
  hipFree(trajectories_d);
  hipFree(res_d);
  delete[] start_points;
  return res;
}
